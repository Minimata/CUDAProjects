#include "hip/hip_runtime.h"
#include "HistogramCustom.h"

#include <iostream>

#include "Device.h"

#include "DataCreator.h"

using std::cout;
using std::endl;

extern __global__ void histogramCustom(int* ptrDevData, int dataMax, int dataSize, int bufferSizeSM);

HistogramCustom::HistogramCustom(const Grid& grid, int dataMax)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    //Data
    this->dataMax = dataMax;
    DataCreator dataCreator(dataMax);
    dataSize = dataCreator.getLength();
    data = dataCreator.getTabData();

    //Threads
    this->nbThreads = grid.threadCounts();  // one dimensionnal block

    //GM
    sizeOctet = dataSize * sizeof(int);

    //SM
    this->sizeSM = dataMax * sizeof(int);
    bufferSizeSM = 128 / sizeof(int);

    //Output
    histogram = nullptr;

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevData, sizeOctet);
	    Device::memclear(ptrDevData, sizeOctet);
	    Device::memcpyHToD(ptrDevData, data, sizeOctet);
	    }

	Device::lastCudaError("Histogram (end allocation)"); // temp debug, facultatif
	}

    }

HistogramCustom::~HistogramCustom(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevData);

	Device::lastCudaError("Histogram MM (end deallocation)"); // temp debug, facultatif
	}
    }

void HistogramCustom::run()
    {
    Device::lastCudaError("Histogram (before)"); // temp debug
    histogramCustom<<<dg, db, sizeSM + bufferSizeSM>>>(ptrDevData, dataMax, dataSize, bufferSizeSM); // assynchrone
    Device::lastCudaError("Histogram (after)"); // temp debug

    int* result = new int[dataMax]();

    Device::memcpyDToH(result, ptrDevData, dataMax*sizeof(int)); // barriere synchronisation implicite
    Device::memclear(ptrDevData, sizeOctet); // On supprime les valeurs laissées en trop

    histogram = result;
    }

void HistogramCustom::display()
    {
    cout << "///////////////////////" << endl;
    cout << "//////TP histogram/////" << endl;
    cout << "///////////////////////" << endl;

    cout << "The Histogram : " << endl;
        for(int i = 0; i < dataMax; i++)
    	{
    	cout << "frequency of " << i << " : " << histogram[i] << endl;
    	}

    cout << "///////////////////////" << endl;
    cout << "//////////END//////////" << endl;
    cout << "///////////////////////" << endl;
    }

bool HistogramCustom::check()
    {
    bool isOk = true;
    int val = 0;
    for (int i = 0; i < dataMax; i++)
	{
	isOk &= (i + 1 == histogram[i]);
	val += histogram[i];
	}

    isOk &= (val == dataSize);
    return isOk;
    }
