#include "hip/hip_runtime.h"
#include "HistogramCustom.h"

#include <iostream>
#include <cmath>

#include "Device.h"

#include "DataCreator.h"

using std::cout;
using std::endl;

extern __global__ void histogramCustom(int** ptrDevData, int dataMax, int dataSizeSM);

HistogramCustom::HistogramCustom(const Grid& grid, int dataMax)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    //Data
    this->dataMax = dataMax;
    DataCreator dataCreator(dataMax);
    dataSize = dataCreator.getLength();
    data = dataCreator.getTabData();

    //Threads
    this->nbThreads = grid.threadCounts();  // one dimensionnal block

    //GM
    sizeOctet = dataSize * sizeof(int);

    //SM
    this->sizeSM = dataMax * sizeof(int);

    //Output

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevData, sizeOctet);
	    Device::memclear(ptrDevData, sizeOctet);
	    //Device::memcpyHToD(ptrDevData, data, sizeOctet);
	    }

	Device::lastCudaError("Histogram (end allocation)"); // temp debug, facultatif
	}

    }

HistogramCustom::~HistogramCustom(void)
    {
    //MM (device free)
	{
	//Device::free (ptrDevNx);

	Device::lastCudaError("Histogram MM (end deallocation)"); // temp debug, facultatif
	}
    }

void HistogramCustom::run()
    {
    Device::lastCudaError("Histogram (before)"); // temp debug
    histogramCustom<<<dg, db, sizeSM + dataSizeSM>>>(ptrDevData, dataMax, dataSizeSM); // assynchrone
    Device::lastCudaError("Histogram (after)"); // temp debug

    Device::memcpyDToH(&histogram, ptrDevData, dataMax*sizeof(int)); // barriere synchronisation implicite
    Device::memclear(ptrDevData, sizeOctet); // On supprime les valeurs laissées en trop
    }

void HistogramCustom::display()
    {
    cout << "///////////////////////" << endl;
    cout << "//////TP histogram/////" << endl;
    cout << "///////////////////////" << endl;

    cout << "///////////////////////" << endl;
    cout << "//////////END//////////" << endl;
    cout << "///////////////////////" << endl;
    }

bool HistogramCustom::check()
    {
    bool isOk = true;
    int val = 0;
    for (int i = 0; i < dataMax; i++)
	{
	isOk &= (i + 1 == histogram[i]);
	val += histogram[i];
	}

    isOk &= (val == dataSize);
    return isOk;
    }
