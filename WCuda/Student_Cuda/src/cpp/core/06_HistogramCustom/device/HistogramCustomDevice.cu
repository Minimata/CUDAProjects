#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <algorithm>

__global__ void histogramCustom(int** ptrDevData, int dataMax, int dataSizeSM);

__device__ void reducIntraThreadCustom(int* tab_SM, int dataSizeSM);
__device__ void reducInterBlockCustom(int* tab_SM, int* tab_GM, int dataSize, int dataMax);
__device__ void copyGMToSMCustom(int* ptrDevTabGM, int* ptrDevTabSM, int dataSizeSM);
__device__ void initSMCustom(int* tab_SM, int dataMax);



__global__ void histogramCustom(int** ptrDevData, int dataMax, int dataSizeSM)
    {
    extern __shared__ int tab_SM[];
    //int tab_GM[] = *(ptrDevData);

    /*
    initSMCustom(tab_SM, dataMax);
    copyGMToSMCustom(tab_GM, tab_SM, dataSizeSM);
    reducIntraThreadCustom (tab_SM, dataSizeSM);
    __syncthreads();
    reducInterBlockCustom(tab_SM, tab_GM, dataSizeSM, dataMax);
    */
    }

__device__ void initSMCustom(int* tab_SM, int dataMax)
    {
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = TID_LOCAL;

    while(s < dataMax)
	{
	tab_SM[s] = 0;
	s+=NB_THREAD_LOCAL;
	}
    }

__device__ void reducIntraThreadCustom(int* tab_SM, int dataSizeSM)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = Indice1D::tidLocal();

    int s = TID;

    while(s < dataSizeSM)
	{
	//atomicAdd(dataSizeSM + tab_SM[s], 1);
	s += NB_THREAD;
	}
    }

__device__ void reducInterBlockCustom(int* tab_SM, int* tab_GM, int dataSizeSM, int dataMax)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = 0;

    while(s < dataMax)
	{
	if(TID < dataMax)
	    {
	    tab_GM[s] = 0;
	    }

	s += NB_THREAD;
	}

    __syncthreads();

    s = 0;

    while(s < dataMax)
	{
	//atomicAdd(tab_GM + s, tab_SM[dataSizeSM + s]);
	s += NB_THREAD_LOCAL;
	}
    }

__device__ void copyGMToSMCustom(int* ptrDevTabGM, int* ptrDevTabSM, int dataSizeSM){

    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    if(TID_LOCAL == 0)
	{
	// memcpy(ptrDevTabSM, ptrDevTabGM, sizeof(int) * dataSizeSM);
	//std::copy(ptrDevTabGM + s, ptrDevTabGM + s + dataSizeSM, ptrDevTabSM);
	}

}

