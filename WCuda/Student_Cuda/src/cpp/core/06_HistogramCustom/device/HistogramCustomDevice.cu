#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"

__global__ void histogramCustom(int* ptrDevData, int dataMax, int dataSize, int bufferSizeSM);

__device__ void reducIntraThreadCustom(int* tab_SM, int* tab_GM, int dataSize, int dataMax, int bufferSizeSM);
__device__ void reducInterBlockCustom(int* tab_SM, int* tab_GM, int dataMax);
__device__ void initSMCustom(int* tab_SM, int dataMax);
__device__ void resetGMCustom(int* tab_GM, int dataMax);

__global__ void histogramCustom(int* ptrDevData, int dataMax, int dataSize, int bufferSizeSM)
    {
    extern __shared__ int tab_SM[];

    initSMCustom(tab_SM, dataMax);
    __syncthreads();
    reducIntraThreadCustom(tab_SM, ptrDevData, dataSize, dataMax, bufferSizeSM);
    __syncthreads();

    resetGMCustom(ptrDevData, dataMax);
    __syncthreads();
    reducInterBlockCustom(tab_SM, ptrDevData, dataMax);
    }

__device__ void initSMCustom(int* tab_SM, int dataMax)
    {
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = TID_LOCAL;

    while (s < dataMax)
	{
	tab_SM[s] = 0;
	s += NB_THREAD_LOCAL;
	}
    }

__device__ void reducIntraThreadCustom(int* tab_SM, int* tab_GM, int dataSize, int dataMax, int bufferSizeSM)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = TID;
    int t = 0;

    //Pattern d'entrelacement sur la GM avec pour unité de copie la transaction
    while (s < dataSize)
	{
	if (TID_LOCAL == 0)
	{
	/** All of these are host methods...
	 *  Apparently memory managment device-side is only int by int, float by float, etc.
	 *  Too bad =(
	 * */

	// hipMalloc(...);
	// hipMemcpy(&tab_SM[dataMax],&tab_GM[s],bufferSizeSM, hipMemcpyDeviceToDevice);
	// memcpy(ptrDevTabSM, ptrDevTabGM, sizeof(int) * dataSizeSM);
	// std::copy(tab_GM + s, tab_GM + s + bufferSizeSM, tab_SM + dataMax);
	}

    __syncthreads();  //I know it's bad

    //Pattern d'entrelacement sur le buffer de la SM
    t = TID_LOCAL;
    while (t < bufferSizeSM)
	{
	atomicAdd(&tab_SM[tab_SM[dataMax + t]], 1);
	t += NB_THREAD_LOCAL;
	}

    s += NB_THREAD;
    }
}

__device__ void resetGMCustom(int* tab_GM, int dataMax)
{
const int TID = Indice1D::tid();
const int NB_THREAD = Indice1D::nbThread();

int s = TID;

while (s < dataMax)
    {
    tab_GM[s] = 0;
    s += NB_THREAD;
    }
}

__device__ void reducInterBlockCustom(int* tab_SM, int* tab_GM, int dataMax)
{
const int TID = Indice1D::tid();
const int NB_THREAD = Indice1D::nbThread();
const int TID_LOCAL = Indice1D::tidLocal();
const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

int s = 0;

while (s < dataMax)
    {
    atomicAdd(&tab_GM[s], tab_SM[s]);
    s += NB_THREAD_LOCAL;
    }
}

