#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>

#include <stdio.h>



__global__ void montecarlo(float* ptrDevNx, int nbSamples, hiprandState* tabDevGenerator);
__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

__device__ void reducIntraThread(float* tab_SM, int nbSamples, hiprandState* tabDevGenerator);
__device__ float fPi(float x);
__device__ int work(float* tab_SM, float x, float y);

/**
 * output : void required !!
 */
__global__ void montecarlo(float* ptrDevNx, int nbSamples, hiprandState* tabDevGenerator)
    {
    extern __shared__ float tab_SM[]; //size defined in calling of kernel

    reducIntraThread(tab_SM, nbSamples, tabDevGenerator);
    __syncthreads();
    reductionADD<float>(tab_SM, ptrDevNx);
    }

__device__ void reducIntraThread(float* tab_SM, int nbSamples, hiprandState* tabDevGenerator)
    {
    const int TID=Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    hiprandState localGenerator = tabDevGenerator[TID];
    float xAlea;
    float yAlea;
    float tmp = 0;
    for (long i = 1; i <= nbSamples; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);

	tmp += work(tab_SM, xAlea, yAlea);
	}

    tab_SM[TID_LOCAL] = tmp;

    tabDevGenerator [TID] = localGenerator;
    }

__device__ int work(float* tab_SM, float x, float y)
    {
    return (int) (fPi(x) >= y);
    }

__device__ float fPi(float x)
    {
    return 1.f / (1.f + x * x);
    }



// Each thread gets same seed, a different sequence number
// no offset
__global__
void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId)
    {
    // Customisation du generator:
    // Proposition, au lecteur de faire mieux !
    // Contrainte : Doit etre différent d'un GPU à l'autre
    // Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGenerator[TID]);
    }
