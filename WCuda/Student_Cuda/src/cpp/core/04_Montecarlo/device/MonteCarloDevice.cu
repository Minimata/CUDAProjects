#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>

#include <stdio.h>

using elem = unsigned int;

__global__ void montecarlo(elem* ptrDevNx, elem nbSamples, hiprandState* tabDevGenerator, float targetHeight, float right, float left, int functionID);
__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

__device__ void reducIntraThread(elem* tab_SM, elem nbSamples, hiprandState* tabDevGenerator, float targetHeight, float right, float left, int functionID);
__device__ elem work(float x, float y, int functionID);
__device__ float map(float x, float left, float right);


__device__ float fPi(float x);
__device__ float identity(float x);
__device__ float squared(float x);


typedef float (*f)(float);

__device__ static f functions[3] = {
	fPi,
	identity,
	squared
};


/**
 * output : void required !!
 */
__global__ void montecarlo(elem* ptrDevNx, elem nbSamples, hiprandState* tabDevGenerator, float targetHeight, float right, float left, int functionID)
    {
    extern __shared__ elem tab_SM[]; //size defined in calling of kernel

    reducIntraThread(tab_SM, nbSamples, tabDevGenerator, targetHeight, right, left, functionID);
    __syncthreads();
    reductionADD<elem>(tab_SM, ptrDevNx);
    }

__device__ void reducIntraThread(elem* tab_SM, elem nbSamples, hiprandState* tabDevGenerator, float targetHeight, float right, float left, int functionID)
    {
    const int TID=Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    hiprandState localGenerator = tabDevGenerator[TID];
    float xAlea;
    float yAlea;
    float tmp = 0;
    for (elem i = 1; i <= nbSamples; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);

	xAlea = map(xAlea, left, right);

	tmp += work(xAlea, targetHeight * yAlea, functionID);
	}

    tab_SM[TID_LOCAL] = tmp;

    tabDevGenerator [TID] = localGenerator;
    }

__device__ elem work(float x, float y, int functionID)
    {
    return (elem) (functions[functionID](x) >= y);
    }

__device__ float map(float x, float left, float right)
    {
    return (right - left) * x + left;
    }

__device__ float squared(float x)
    {
    return x*x;
    }

__device__ float identity(float x)
    {
    return x;
    }

__device__ float fPi(float x)
    {
    return 1.f / (1.f + x * x);
    }



// Each thread gets same seed, a different sequence number
// no offset
__global__
void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId)
    {
    // Customisation du generator:
    // Proposition, au lecteur de faire mieux !
    // Contrainte : Doit etre différent d'un GPU à l'autre
    // Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGenerator[TID]);
    }
