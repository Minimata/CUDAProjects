#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include <cmath>
#include "MonteCarlo.h"

using std::cout;
using std::endl;


extern __global__ void montecarlo(float* ptrDevNx, int nbmontecarlo, hiprandState* ptrDevCurand);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

#define PI 3.14159265358979323846264338327950288419716939937510


MonteCarlo::MonteCarlo(const Grid& grid, int nbSamplesPower, float targetHeight, float tolerance)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    this->nbThreads = grid.threadCounts();  // one dimensionnal block

    this->sizeSM = this->db.x * sizeof(float);  // size of SM tab
    this->nbSamples = pow(2, nbSamplesPower);
    this->targetHeight = targetHeight;
    this->tolerance = tolerance;
    this->pi = 0;

    size_t sizeCurand = this->nbThreads * sizeof(hiprandState) ;
    this->ptrDevCurand=NULL;

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevNx, sizeof(float));
	    Device::memclear(ptrDevNx, sizeof(float));

	    Device::malloc(&ptrDevCurand, sizeCurand);
	    Device::memclear(ptrDevCurand, sizeof(float));
	    }

	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug, facultatif
	}

    }

MonteCarlo::~MonteCarlo(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevNx);
	Device::free(ptrDevCurand);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }


float MonteCarlo::run()
    {
    int nbSamplesPerThread = nbSamples / nbThreads;
    Device::lastCudaError("hiprand (before)"); // temp debug
    setup_kernel_rand<<<dg, db>>>(ptrDevCurand, 0);
    Device::lastCudaError("montecarlo (before)"); // temp debug
    montecarlo<<<dg, db, sizeSM>>>(ptrDevNx, nbSamplesPerThread, ptrDevCurand); // assynchrone
    Device::lastCudaError("montecarlo (after)"); // temp debug

    Device::memcpyDToH(&pi, ptrDevNx, sizeof(float)); // barriere synchronisation implicite
    //pi prend la valeur de Nx ici


    cout << "Nx : " << pi << endl;
    cout << "target height : " << this->targetHeight << endl;
    cout << "nb Samples : " << this->nbSamples << endl;
    cout << "samples per thread : " << nbSamplesPerThread << endl;

    pi *= this->targetHeight / this->nbSamples;

    return pi;
    }

void MonteCarlo::display()
    {
    cout << "///////////////////////" << endl;
    cout << "/////TP montecarlo/////" << endl;
    cout << "///////////////////////" << endl;

    cout << "Exact Pi : \t" << PI << endl;
    cout << "Estimation : \t" << this->pi << endl;

    cout << "///////////////////////" << endl;
    cout << "//////////END//////////" << endl;
    cout << "///////////////////////" << endl;
    }

bool MonteCarlo::check()
    {
    return std::fabs(this->pi - PI) < this->tolerance;
    }
