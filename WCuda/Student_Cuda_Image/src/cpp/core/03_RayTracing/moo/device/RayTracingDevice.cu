#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RayTracingMath.h"
using namespace gpu;


__global__ void raytracing(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels,uint w, uint h, float t);


__global__ void raytracing(Sphere* ptrDevTabSphere,int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(nbSphere, ptrDevTabSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i, j;

    int s = TID;
    while (s < w*h)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	rayTracingMath.colorIJ(&s[ptrDevPixels],i, j, t);

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

