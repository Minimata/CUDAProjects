#include "hip/hip_runtime.h"
#include "RayTracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

#include "Sphere.h"
#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt, int nbSphere) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    this->nbSphere = nbSphere;

    this->sizeOctet=nbSphere*sizeof(Sphere);

    SphereCreator sphereCreator(nbSphere, w, h); // sur la pile
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();
    // transfert to GM
    toGM(ptrTabSphere);
    // transfert to CM
    // toCM(ptrTabSphere); // a implemneter
    // transfert to SM
    // toSM(ptrTabSphere); // a implemneter

    }

RayTracing::~RayTracing()
{
    Device::free(ptrDevTabSphere);
}

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

void RayTracing::toGM(Sphere* ptrTabSphere)
{
    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

}
void RayTracing::toCM(Sphere* ptrTabPixel)
{

}
void RayTracing::toSM(Sphere* ptrTabPixel)
{

}

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
{
Device::lastCudaError("raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

raytracing<<<dg,db>>>(this->ptrDevTabSphere, this->nbSphere, ptrDevPixels, w, h, t);

Device::lastCudaError("raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
}

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
{
    t += dt;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

