#include "hip/hip_runtime.h"
#include "RayTracing.h"
#include "length_cm.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

#include "Sphere.h"
#include "SphereCreator.h"

using std::cout;
using std::endl;

extern __global__ void raytracing_GM(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t);
extern __global__ void raytracing_CM(uchar4* ptrDevPixels, uint w, uint h, float t);
extern __global__ void raytracing_SM(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t);

extern __host__ void uploadGPU(Sphere* tabValue);

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt, int nbSphere) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    this->nbSphere = nbSphere;

    this->sizeOctet = nbSphere * sizeof(Sphere);

    SphereCreator sphereCreator(nbSphere, w, h); // sur la pile
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();
    // transfert to GM
    toGM(ptrTabSphere);
    // transfert to CM
    toCM(ptrTabSphere);

    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

void RayTracing::toGM(Sphere* ptrTabSphere)
    {
    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

    }
void RayTracing::toCM(Sphere* ptrTabPixel)
    {
    uploadGPU(ptrTabPixel);
    }

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    static int i = 0;
    if (i % 3 == 0)
	{
	 raytracing_GM<<<dg,db>>>(this->ptrDevTabSphere, this->nbSphere, ptrDevPixels, w, h, t);
	}
    else if (i % 3 == 1)
	{
	 raytracing_CM<<<dg,db>>>(ptrDevPixels, w, h, t);
	}
    else if (i % 3 == 2)
	{
	 raytracing_SM<<<dg,db, this->sizeOctet>>>(this->ptrDevTabSphere, this->nbSphere, ptrDevPixels, w, h, t);
	}
    i++;

    Device::lastCudaError("raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
{
    t += dt;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

