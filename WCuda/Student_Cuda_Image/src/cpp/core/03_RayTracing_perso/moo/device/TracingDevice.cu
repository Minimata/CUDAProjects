#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "TracingMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void tracing(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void tracing(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    TracingMath tracingMath = TracingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i, j = 0;
    while(s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	tracingMath.colorIJ(&s[ptrDevPixels], i, j, t);  // s[ptrTabPixels] == ptrTabPixels[s] => true

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

