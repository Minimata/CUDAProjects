#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include "Device.h"
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath);

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt, const DomaineMath& domaineMath) :
	Fractal(grid, w, h, dt, domaineMath)
    {

    }

Mandelbrot::~Mandelbrot() {}

void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath) {

    mandelbrot<<<dg, db>>>(ptrDevPixels, w, h, this->t, domaineMath);
}

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

