#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "FractalMath.h"
#include "MandelbrotMath.h"
#include "JuliaMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;



__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath);
__global__ void julia(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath);
__device__ void process(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, FractalMath* fractalMath);
__device__ void workPixel(uchar4* ptrColorIJ, int i, int j, DomaineMath domaineMath, FractalMath* fractalMath);



__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath) {
    FractalMath* fractal = new MandelbrotMath(t);
    process(ptrDevPixels, w, h, domaineMath, fractal);
    delete fractal;
}

__global__ void julia(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath) {
    FractalMath* fractal = new JuliaMath(t);
    process(ptrDevPixels, w, h, domaineMath, fractal);
    delete fractal;
}

__device__ void process(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, FractalMath* fractalMath)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int s = TID;
    int i = 0, j = 0;
    while(s < w*h)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	workPixel(&ptrDevPixels[s], i, j, domaineMath, fractalMath);
	s += NB_THREAD;
	}
    }

__device__ void workPixel(uchar4* ptrColorIJ, int i, int j, DomaineMath domaineMath, FractalMath* fractalMath)
    {
	double x;
	double y;
	domaineMath.toXY(i, j, &x, &y);
	fractalMath->colorXY(ptrColorIJ, x, y);
    }
